
#include <hip/hip_runtime.h>



template <class value_type> class nonlinearfit{

	typedef value_type (*functions)(value_type, value_type*);




	public:

	//Number of data points
	  unsigned int n;

	//Number of params
	  unsigned int p;

	//Function to be fitted
	  value_type (*F)(value_type, value_type*);

	//Jacobian functions used for fit
	  functions * Jacobianfunctions;

	  value_type mu1Inc;
	  value_type mu2Inc;

	  value_type  lambda1;

	  value_type  lambda2;

	  value_type  mu1;

	  value_type  mu2;




	//Y values
	  value_type * Y;

	//X values
	  value_type * X;

	  value_type * B;

	  value_type * delta;



	  nonlinearfit(unsigned int nD, unsigned int  nP){
		  n = nD;
		  p = nP;




		  Jacobianfunctions = new functions[nP];

		  X = new value_type[nD];
		  Y = new value_type[nD];
		  B = new value_type[nP];

		  delta = new value_type[nP];


	  }
	  ~nonlinearfit(){
		  delete [] X;
		  delete [] B;
		  delete [] delta;
		  delete Jacobianfunctions;


	  }
	  void setInitialIterIncr(value_type lambda1Ini, value_type lambda2Ini,
			  value_type mu1Ini, value_type mu2Ini, value_type mu1Incn, value_type mu2Incn){


		  lambda1 = lambda1Ini;
		  lambda2 = lambda2Ini;

		  mu1 = mu1Ini;
		  mu2 = mu2Ini;
		  mu1Inc = mu1Incn;
		  mu2Inc = mu2Incn;

	  }

	  value_type calcChi(value_type * rB){
		  value_type sum = 0.f;
		  for(int i = 0 ; i < n; i++){
			  sum += Y[i]*Y[i] - 2.f*Y[i]*F(X[i], rB) + F(X[i], rB)*F(X[i], rB);
		  }
		  return sum;

	  }

	  bool solveIter(){



		  value_type **JTJ;
		  value_type * JTY;
		  JTY = new value_type[p];
		  JTJ = new value_type*[p];
		  for(int i = 0 ; i < p ; i++){
			  JTJ[i] = new value_type[p];
		  }

		  generateMatrix(JTJ, JTY);

		  solveMatrix(JTJ, JTY, delta);
		  delete [] JTJ;
		  delete [] JTY;
		  value_type iniChi = calcChi(B);

		  value_type * nB = new value_type[p];
		  for(int m = 0 ; m < p ; m++){
			  nB[m] = B[m] + delta[m];
		  }

		  value_type newChi = calcChi(nB);

		  if(newChi < iniChi){
			  lambda1 = lambda1*mu1;
			  lambda2 = lambda2*mu2;
			  mu1 = mu1*mu1Inc;
			  mu2 = mu2*mu2Inc;

			  for(int m = 0 ; m < p ; m++){
				  B[m] = nB[m];

			  }

			  delete [] nB;
			  return true;
		  }else{

			  lambda1 = lambda1/mu1;
			  lambda2 = lambda2/mu2;
			  mu1 = mu1Inc;
			  mu2 = mu2Inc;
			  delete [] nB;
			  return false;
		  }

	  }

	  bool solve(value_type precision, int max){
		  int iter = 0;
		  bool success;
		  do{
			  success = solveIter();
			  iter += 1;
		  }while((iter < max) && (calcChi(B) > precision));

		  return success;
	  }



	private:

	  void solveMatrix(value_type ** A, value_type * b, value_type * x){
			float L[p][p];
			float U[p][p];



			for(int r = 0; r < p; r++){
				for(int c = 0 ; c < p ; c++){
					if(r >= c){
					if(c == r){
						L[r][c] = 1.f;
					}else{
						float sum = A[r][c];
						for(int s = 0; s < c; s++){
							sum += -L[r][s]*U[s][c];
						}
						sum = sum/U[c][c];
						L[r][c] = sum;
					}
					}else{
						L[r][c] = 0.f;
					}
					if(c >= r){
						float sum = A[r][c];
						for(int s = 0 ; s < r ; s++){
							sum += -L[r][s]*U[s][c];
						}

						U[r][c] = sum;
					}else{
						U[r][c] = 0.f;
					}

				}

			}
			float y[3];

			for(int r = 0; r < p; r++){
				float sum = b[r];
				for(int c = 0; c < r ; c++){
					sum += -y[c]*L[r][c];
				}
				y[r] = sum;
			}

			for(int r = p - 1; r >= 0 ; r = r - 1){
				float sum = y[r];
				for(int c = p - 1 ; c > r  ; c = c - 1){
					sum += -x[c]*U[r][c];
				}
				sum = sum/U[r][r];
				x[r] = sum;
			}
	  }


	  void generateMatrix(value_type ** A, value_type * b){
		  for(int r = 0 ; r < p ; r++){
			  for(int c = 0 ; c < p ; c++){
				  value_type JTJsum = 0.f;
				  value_type JTYsum = 0.f;
				  for(int w = 0; w < n; w++){
					  if(r == 0){
						  JTYsum += (Y[w] - F(X[w], B))*Jacobianfunctions[c](X[w], B);
					  }
					  JTJsum += Jacobianfunctions[c](X[w], B)*Jacobianfunctions[r](X[w], B);
				  }

				  if(r == 0){
					  b[c] = lambda2*JTYsum;
				  }
				  A[r][c] = JTJsum;
			  }

		  }

		  A[0][0] += A[0][0]*lambda1;
		  A[1][1] += A[1][1]*lambda1;
		  A[2][2] += A[2][2]*lambda1;
	  }







};







